#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define ull unsigned long long

__device__ void ithpermutation(ull n, ull i, ull *fact, ull *path)
{
    long long j, k = 0;
    for (k = 0; k < n; ++k)
    {
        path[k] = i / fact[n - 1 - k];
        i = i % fact[n - 1 - k];
    }

    // Adjust values to obtain the permutation
    for (k = n - 1; k > 0; --k)
        for (j = k - 1; j >= 0; --j)
            if (path[j] <= path[k])
                path[k]++;
}
__device__ int lock_me(int *mutex, int id)
{
    // Atomic Compare and Swap
    if (atomicCAS((int *)(mutex + id), 0, 1) == 0)
        return 1;
    return 0;
}

__device__ void unlock_me(int *mutex, int id)
{

    atomicExch((int *)(mutex + id), 0);
}

// Function to read the graph from a file
__host__ void readGraphFromFile(FILE *file, double *graph, int N)
{
    if (file == NULL)
    {
        fprintf(stderr, "Error opening file\n");
        exit(EXIT_FAILURE);
    }

    int u, v;
    double w;
    while (fscanf(file, "%d %d %lf", &u, &v, &w) == 3)
    {
        graph[u * N + v] = w;
        graph[v * N + u] = w;
    }

    fclose(file);
}
// Function to calculate the factorial array
__host__ void calculateFactorials(ull n, ull *fact)
{
    fact[0] = 1;
    for (ull i = 1; i <= n; ++i)
    {
        fact[i] = fact[i - 1] * i;
    }
}

__global__ void calculatePathWeight(double *graph, ull *fact, double *min_path, int *best_path, int *lock, ull numPerms, ull numVertices)
{
    ull idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPerms)
        return;

    // prepare shared memory
    extern __shared__ ull shared_array[];
    ull *shared_factorial = (ull *)&shared_array[0];
    double *shared_graph = (double *)&shared_array[numVertices + 1];

    int fact_load_thread = (numVertices + 1 + blockDim.x - 1) / blockDim.x;
    for (int i = 0; i < fact_load_thread; i++)
    {
        if (threadIdx.x * fact_load_thread + i <= numVertices)
        {
            shared_factorial[threadIdx.x * fact_load_thread + i] = fact[threadIdx.x * fact_load_thread + i];
        }
    }
    int graph_elements = numVertices * numVertices;
    int graph_load_thread = (graph_elements + blockDim.x - 1) / blockDim.x;
    for (int i = 0; i < graph_load_thread; i++)
    {
        if (threadIdx.x * graph_load_thread + i <= graph_elements)
        {
            shared_graph[threadIdx.x * graph_load_thread + i] = graph[threadIdx.x * graph_load_thread + i];
        }
    }
    __syncthreads();

    ull total_threads = blockDim.x * gridDim.x;
    ull permutatins_per_thread = (numPerms + total_threads - 1) / total_threads;
    ull *path = new ull[numVertices];

    ull *best_path_thread = new ull[numVertices];
    double best_cost_thread = min_path[0];
    for (ull path_num = idx * permutatins_per_thread; path_num < (idx + 1) * permutatins_per_thread; path_num++)
    {
        if (path_num >= numPerms)
            break;

        ithpermutation(numVertices, path_num, shared_factorial, path);
        double current_pathweight = 0;
        for (ull j = 1; j < numVertices; j++)
        {
            ull u = path[j - 1];
            ull v = path[j];
            current_pathweight += shared_graph[u * numVertices + v];
        }
        ull s = path[0];
        ull k = path[numVertices - 1];
        current_pathweight += shared_graph[k * numVertices + s];

        if (current_pathweight < best_cost_thread)
        {
            best_cost_thread = current_pathweight;
            for (int i = 0; i < numVertices; i++)
            {
                best_path_thread[i] = path[i];
            }
        }
    }

    int successfull = 0;
    while (!successfull)
    {
        if (lock_me(lock, 0))
        { // lock acquired?
            successfull = 1;
        }
    }

    if (best_cost_thread < min_path[0])
    {
        min_path[0] = best_cost_thread;
        for (int i = 0; i < numVertices; i++)
        {
            best_path[i] = best_path_thread[i];
        }
    }

    unlock_me(lock, 0);

    delete[] path;
}

int main(int argc, char **argv)
{

    char *file_name;
    if (argc != 2)
    {
        fprintf(stderr, "Usage: %s <filename>\n", argv[0]);
        return EXIT_FAILURE;
    }
    file_name = argv[1];
    // file_name="edges.txt";
    FILE *file = fopen(file_name, "r");
    if (file == NULL)
    {
        fprintf(stderr, "Error opening file\n");
        exit(EXIT_FAILURE);
    }
    int N;
    fscanf(file, "%d", &N);

    // Read the graph from the file

    double *h_graph;
    ull *h_fact;
    int *h_best_path;
    double h_min_path = DBL_MAX;
    int h_lock = 0;

    // allocate the host memory using cudaMallocHost, to allocate it in the pinned memory which is faster to access
    // pinned memory is faster to access than pageable memory because it is directly accessible from the direct memory access (DMA) engine of the GPU
    // The only difference is that the allocated memory cannot be paged by the OS
    hipHostMalloc((void **)&h_fact, (N + 1) * sizeof(ull), hipHostMallocDefault);
    hipHostMalloc((void **)&h_best_path, N * sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void **)&h_graph, N * N * sizeof(double), hipHostMallocDefault);
    hipDeviceSynchronize();
    readGraphFromFile(file, h_graph, N);
    calculateFactorials((ull)N, h_fact);

    double *d_graph;
    ull *d_fact;
    double *d_min_path;
    int *d_best_path;
    int *d_lock;

    hipMalloc(&d_best_path, N * sizeof(int));
    hipMalloc(&d_graph, N * N * sizeof(double));
    hipMalloc(&d_fact, (N + 1) * sizeof(ull));
    hipMalloc(&d_min_path, sizeof(double));
    hipMalloc(&d_lock, sizeof(int));

    hipMemcpy(d_graph, h_graph, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_fact, h_fact, (N + 1) * sizeof(ull), hipMemcpyHostToDevice);
    hipMemcpy(d_min_path, &h_min_path, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_lock, &h_lock, sizeof(int), hipMemcpyHostToDevice);

    ull numPerms = h_fact[N];
    ull blockSize = 1024;
    ull numBlocks = 15;

    calculatePathWeight<<<numBlocks, blockSize, N * N * sizeof(double) + (N + 1) * sizeof(ull)>>>(d_graph, d_fact, d_min_path, d_best_path, d_lock, numPerms, (ull)N);

    hipMemcpy(h_best_path, d_best_path, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_min_path, d_min_path, sizeof(double), hipMemcpyDeviceToHost);
    printf("\n\nMinimum Path Weight: %lf\n\n", h_min_path);
    printf("Best Path: ");
    for (int i = 0; i < N; i++)
    {
        printf("%d ", h_best_path[i]);
    }
    printf("\n");
    hipFree(d_best_path);
    hipFree(d_graph);
    hipFree(d_fact);
    hipFree(d_min_path);
    hipHostFree(h_graph);
    hipHostFree(h_fact);
    hipHostFree(h_best_path);

    return 0;
}
