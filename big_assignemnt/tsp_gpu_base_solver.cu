#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define ull unsigned long long

__device__ void ithpermutation(ull n,ull i,ull* fact,ull*path){
        long long j, k = 0;
        for (k = 0; k < n; ++k) {
            path[k] = i / fact[n - 1 - k];
            i = i % fact[n - 1 - k];
        }

        // Adjust values to obtain the permutation
        for (k = n - 1; k > 0; --k)
            for (j = k - 1; j >= 0; --j)
                if (path[j] <= path[k])
                    path[k]++;

}
// __device__ void lock_me(int* mutex, int id) {

//           while (atomicCAS((int*) (mutex + id), 0, 1) != 0);
// }


__device__ int lock_me(int* mutex, int id) {

if (atomicCAS((int*) (mutex + id), 0, 1) == 0)
  return 1;
return 0;
}


__device__ void unlock_me(int* mutex, int id) {
    atomicExch((int*) (mutex + id), 0);
}

__global__ void calculatePathWeight(double *graph, ull *fact, double *min_path,int* best_path,int *lock, ull numPerms, ull numVertices) {
    ull idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= numPerms) return;

    ull *path = new ull[numVertices];
    ithpermutation(numVertices,idx,fact,path);
    double current_pathweight = 0;
    for (ull j = 1; j < numVertices; j++) {
        ull u = path[j - 1];
        ull v = path[j];
        current_pathweight += graph[u * numVertices + v];
    }
    ull s = path[0];
    ull k = path[numVertices - 1];
    current_pathweight += graph[k * numVertices + s];
    
    
    // while (atomicExch(lock, 1) != 0);
    // while (atomicCAS(lock, 0, 1) != 0);
    // lock =1;
    // lock_me(lock,0);
    int successfull = 0;
    while (!successfull){
    if (lock_me(lock, 0)) { //lock acquired?
        unlock_me(lock, 0); // then unlock
        successfull = 1;}
    }



    if(current_pathweight<min_path[0]){
        min_path[0] = current_pathweight;
        for (int i = 0; i < numVertices; i++) {
            best_path[i] = path[i];
        }
    }
    unlock_me(lock,0);
    // lock=0;
    // atomicExch(lock, 0);
    //atomicMin_double(min_path, current_pathweight);

    delete[] path;
}
// Function to read the graph from a file
__host__ void readGraphFromFile(FILE *file, double** graph) {
    if (file == NULL) {
        fprintf(stderr, "Error opening file\n");
        exit(EXIT_FAILURE);
    }

    int u, v;
    double w;
    while (fscanf(file, "%d %d %lf", &u, &v, &w) == 3) {
        graph[u][v] = w;
        graph[v][u] = w;
    }

    fclose(file);
}
// Function to calculate the factorial array
__host__ void calculateFactorials(ull n, ull* fact) {
    fact[0] = 1;
    for (ull i = 1; i <= n; ++i) {
        fact[i] = fact[i - 1] * i;
    }
}

int main(int argc, char **argv) {
    char*file_name;
    if (argc != 2) {
        fprintf(stderr, "Usage: %s <filename>\n", argv[0]);
        return EXIT_FAILURE;
    }
    file_name = argv[1];
    // file_name="edges.txt";
    FILE *file = fopen(file_name, "r");
    if (file == NULL) {
        fprintf(stderr, "Error opening file\n");
        exit(EXIT_FAILURE);
    }
    int N;
    fscanf(file,"%d",&N);
    double** graph = (double**)malloc(N * sizeof(double*));
    for(int i=0;i<N;i++){
        graph[i]=(double*)malloc(N * sizeof(double));
    }
    // Initialize the graph with 0s for diagonal and infinity for other entries
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (i == j) {
                graph[i][j] = 0;
            } else {
                graph[i][j] = DBL_MAX;
            }
        }
    }
    // Read the graph from the file
    readGraphFromFile(file, graph);

    double *h_graph = (double *)malloc(N * N * sizeof(double));
    double *d_graph;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            h_graph[i * N + j] = graph[i][j];
        }
    }
    hipMalloc(&d_graph, N * N * sizeof(double));
    hipMemcpy(d_graph, h_graph, N * N * sizeof(double), hipMemcpyHostToDevice);

    ull *h_fact = (ull *)malloc((N + 1) * sizeof(ull));
    ull *d_fact;
    calculateFactorials((ull)N, h_fact);
    hipMalloc(&d_fact, (N + 1) * sizeof(ull));
    hipMemcpy(d_fact, h_fact, (N + 1) * sizeof(ull), hipMemcpyHostToDevice);

    double *d_min_path;
    hipMalloc(&d_min_path, sizeof(double));
    double h_min_path = DBL_MAX;
    hipMemcpy(d_min_path, &h_min_path, sizeof(double), hipMemcpyHostToDevice);

    int *h_best_path = (int *)malloc(N * sizeof(int));
    int *d_best_path;
    hipMalloc(&d_best_path, N * sizeof(int));
    
    int h_lock = 0;
    int *d_lock;
    hipMalloc(&d_lock, sizeof(int));
    hipMemcpy(d_lock, &h_lock, sizeof(int), hipMemcpyHostToDevice);

    ull numPerms = h_fact[N];
    ull blockSize = 16;
    ull numBlocks = (numPerms + blockSize - 1) / blockSize;
    calculatePathWeight<<<numBlocks, blockSize>>>(d_graph, d_fact, d_min_path,d_best_path,d_lock, numPerms,(ull) N);

    hipMemcpy(h_best_path, d_best_path, N * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&h_min_path, d_min_path, sizeof(double), hipMemcpyDeviceToHost);
    printf("\n\nMinimum Path Weight: %lf\n\n", h_min_path);
    printf("Best Path: ");
    for (int i = 0; i < N; i++) {
        printf("%d ", h_best_path[i]);
    }
    printf("\n");
    hipFree(d_best_path);
    hipFree(d_graph);
    hipFree(d_fact);
    hipFree(d_min_path);
    free(h_graph);
    free(h_fact);
    free(h_best_path);

    return 0;
}
