/**
    Ahmed_Hany_Farouk_1_10

    Sec:    1
    BN:     10
    Code:   9202213

    Ahmed_Sayed_Sayed_1_3

    Sec:    1
    BN:     3
    Code:   9202111
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

/**
 * @brief Perform matrix addition
 * @param C : Resultant matrix (output)
 * @param A : First input matrix
 * @param B : Second input matrix
 * @param n : Number of rows
 * @param m : Number of columns
 */
 __global__ void k3(float *C, float *A, float *B, int n, int m) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < m) {
        for (int i = 0; i < n; ++i) {
            C[i * m + col] = A[i * m + col] + B[i * m + col];
        }
    }
}

int main(int argc, char *argv[]) {

    // Check if the correct number of command-line arguments is provided
    if (argc != 3) {
        fprintf(stderr, "You should use: %s input_filename output_filename\n", argv[0]);
        return 1;
    }

    // Open the input file and check for errors
    FILE *input_file = fopen(argv[1], "r");
    if (!input_file) {
        fprintf(stderr, "Error: Cannot open input file %s\n", argv[1]);
        return 1;
    }

    // Open the output file and check for errors
    FILE *output_file = fopen(argv[2], "w");
    if (!output_file) {
        fprintf(stderr, "Error: Cannot open output file %s\n", argv[2]);
        fclose(input_file);
        return 1;
    }

    int t, n, m;
    float *matrix1, *matrix2, *out;
    float *d_matrix1, *d_matrix2, *d_out;
    
    // Read the number of test cases from the input file
    fscanf(input_file, "%d", &t);
    for (int tt = 0; tt < t; tt++) {

        // Read the number of rows and columns for the current test case
        fscanf(input_file, "%d%d", &n, &m);

        // Allocate memory for matrices on the host
        matrix1 = (float*)malloc(sizeof(float) * n * m);
        matrix2 = (float*)malloc(sizeof(float) * n * m);
        out = (float*)malloc(sizeof(float) * n * m);

        // Read the values of matrix1 from the input file
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < m; j++) {
                fscanf(input_file, "%f", &matrix1[i * m + j]);
            }
        }

        // Read the values of matrix2 from the input file
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < m; j++) {
                fscanf(input_file, "%f", &matrix2[i * m + j]);
            }
        }

        // Allocate memory for matrices on the device
        hipMalloc((void**)&d_matrix1, n * m * sizeof(float));
        hipMalloc((void**)&d_matrix2, n * m * sizeof(float));
        hipMalloc((void**)&d_out, n * m * sizeof(float));

        // Copy data from host to device
        hipMemcpy(d_matrix1, matrix1, n * m * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_matrix2, matrix2, n * m * sizeof(float), hipMemcpyHostToDevice);

        // Define the number of threads per block
        int threadsPerBlock = 256;

        // Define the number of blocks
        int numBlocks = (m + threadsPerBlock - 1) / threadsPerBlock;

        // Launch kernel
        k3<<<numBlocks, threadsPerBlock>>>(d_out, d_matrix1, d_matrix2, n, m);
        
        // Copy result back to host
        hipMemcpy(out, d_out, sizeof(float) * n * m, hipMemcpyDeviceToHost);
        
        // Write the result to the output file
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < m; j++) {
                fprintf(output_file, "%f ", out[i * m + j]);
            }
            fprintf(output_file, "\n");
        }

        // Free device memory
        hipFree(d_matrix1);
        hipFree(d_matrix2);
        hipFree(d_out);

        // Free host memory
        free(matrix1);
        free(matrix2);
        free(out);
    }

    // Close input and output files
    fclose(input_file);
    fclose(output_file);

    return 0;
}
